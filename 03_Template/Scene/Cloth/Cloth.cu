
#include <hip/hip_runtime.h>

#define TIME_STEPSIZE2 0.3

__global__ void addforce_Kernel(float3* partcleAcclerationArray, float3 accelaration)
{
	int thread_no = blockIdx.x * blockDim.x + threadIdx.x;
	
	partcleAcclerationArray[thread_no] = make_float3(accelaration.x , accelaration.y , accelaration.z );
}


void addforceCudaKernel(float3* partcleAcclerationArray, float3 accelaration, int numOfParticles,int particle_width, int particle_height)
{
 
  dim3 DimBlock=dim3(particle_width ,1,1);
  dim3 DimGrid=dim3(particle_height, 1, 1);
 
  addforce_Kernel<<<DimGrid, DimBlock>>>(partcleAcclerationArray, accelaration);
  
}



__device__ float3 GPUgetParticle(float3* GPUParticles_pos_array,int x, int y, int particle_width) { return GPUParticles_pos_array[y*particle_width + x]; }

__device__ float GPUcrossProduct[3];

__device__ float* GPUcross(const float* v1, const float* v2)
{
	float crossProduct[3] = {v1[1] * v2[2] - v1[2] * v2[1], v1[2] * v2[0] - v1[0] * v2[2], v1[0] * v2[1] - v1[1] * v2[0]};
	return crossProduct;
}

__device__ float GPUlength(float* v)
{
	return sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
}

__device__ int GPUgetParticles_pos_array_index(int x, int y, int particle_width) 
{ 
	return (y*particle_width + x); 
}

__global__ void applyWindForce_Kernel(float3* GPUParticles_pos_array, float3* partcleAcclerationArray,int particle_width,int particle_height,  float3 windForce)
{

	int thread_no = blockIdx.x * blockDim.x + threadIdx.x;
	
	float *normal;
	float3 TrianglePoint1, TrianglePoint2, TrianglePoint3;
	float v1[3], v2[3];
	float mass = 1.0;
	if(((thread_no) < ((particle_width * particle_height) - blockDim.x  )) && ((thread_no % blockDim.x) < (blockDim.x-1)))
	{
		
		TrianglePoint1 = GPUParticles_pos_array[thread_no + blockDim.x];//  ,X+1,Y,particle_width;
		TrianglePoint2 = GPUParticles_pos_array[thread_no];//  ,X,Y,particle_width;
		TrianglePoint3 = GPUParticles_pos_array[thread_no + 1];// ,X,Y+1,particle_width;
		
		
		v1[0] = TrianglePoint2.x - TrianglePoint1.x;
		v1[1] = TrianglePoint2.y - TrianglePoint1.y;
		v1[2] = TrianglePoint2.z - TrianglePoint1.z;
				
		v2[0] = TrianglePoint3.x - TrianglePoint1.x;
		v2[1] = TrianglePoint3.y - TrianglePoint1.y;
		v2[2] = TrianglePoint3.z - TrianglePoint1.z;
		
		normal = GPUcross(&v1[0], &v2[0]);
		
		float d[3];
		float l = GPUlength(&normal[0]);
		
		d[0] = normal[0]/l;
		d[1] = normal[1]/l;
		d[2] = normal[2]/l;	
		
		float dotproduct = d[0] * windForce.x + d[1] * windForce.y + d[2] * windForce.z;
		float force[3] = {normal[0] * dotproduct,normal[1] * dotproduct,normal[1] * dotproduct} ;
		
		int particalIndex = 0;
			particalIndex = thread_no + blockDim.x ; //GPUgetParticles_pos_array_index(X+1,Y,particle_width);
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
			
			
			particalIndex = thread_no;// GPUgetParticles_pos_array_index(X,Y,particle_width);
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
			
			
			particalIndex = thread_no+1;//GPUgetParticles_pos_array_index(X,Y+1,particle_width);
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
			
			
			
			TrianglePoint1 = GPUParticles_pos_array[thread_no + blockDim.x + 1] ;//,X+1,Y+1,particle_width);
			TrianglePoint2 = GPUParticles_pos_array[thread_no+ blockDim.x ];  //,X+1,Y,particle_width);
			TrianglePoint3 = GPUParticles_pos_array[thread_no + 1]; //,X,Y+1,particle_width);
			
			
			v1[0] = TrianglePoint2.x - TrianglePoint1.x;
			v1[1] = TrianglePoint2.y - TrianglePoint1.y;
			v1[2] = TrianglePoint2.z - TrianglePoint1.z;
			
			v2[0] = TrianglePoint3.x - TrianglePoint1.x;
			v2[1] = TrianglePoint3.y - TrianglePoint1.y;
			v2[2] = TrianglePoint3.z - TrianglePoint1.z;
			
			normal = GPUcross(&v1[0], &v2[0]);
			
			l = GPUlength(&normal[0]);
			d[0] = normal[0]/l;
			d[1] = normal[1]/l;
			d[2] = normal[2]/l;			
			
			dotproduct = d[0] * windForce.x + d[1] * windForce.y + d[2] * windForce.z;
			force[0] = normal[0] * dotproduct; force[1] = normal[1] * dotproduct; force[2] = normal[1] * dotproduct;
			
			
			
			particalIndex = thread_no + blockDim.x + 1;//GPUgetParticles_pos_array_index(X+1,Y+1,particle_width);
			
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
			
			
			particalIndex = thread_no + blockDim.x ;//GPUgetParticles_pos_array_index(X+1,Y,particle_width);
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
			
			
			particalIndex = thread_no+1;//GPUgetParticles_pos_array_index(X,Y+1,particle_width);
			
			partcleAcclerationArray[particalIndex].x = partcleAcclerationArray[particalIndex].x + force[0] / mass;
			partcleAcclerationArray[particalIndex].y = partcleAcclerationArray[particalIndex].y + force[1] / mass;
			partcleAcclerationArray[particalIndex].z = partcleAcclerationArray[particalIndex].z + force[2] / mass;
	}	
	
}


void applyWindForceCudaKernel(float3* GPUParticles_pos_array, float3* partcleAcclerationArray, int particle_width, int particle_height, float3 windForce)
{
	dim3 DimBlock=dim3(particle_width ,1,1);
	dim3 DimGrid=dim3( particle_height,1, 1);
	
	applyWindForce_Kernel<<<DimGrid, DimBlock>>>(GPUParticles_pos_array,partcleAcclerationArray,particle_width,particle_height,windForce);
}


__global__ void timeStep_Kernel(float3* GPUParticles_pos_array,int2* GPUNeighbourParticlesInddex,float* GPURestDistance,bool* GPUMovableStatus,int TotalThreads)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < TotalThreads)
	{

		float p1_to_p2[3];
		p1_to_p2[0] = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].x - GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].x;
		p1_to_p2[1] = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].y - GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].y;
		p1_to_p2[2] = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].z - GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].z;
												
		float current_distance = GPUlength(&p1_to_p2[0]); // current distance between p1 and p2
				
					
		float correctionVector[3];
		correctionVector[0] = p1_to_p2[0] * (1 - GPURestDistance[index] / current_distance);
		correctionVector[1] = p1_to_p2[1] * (1 - GPURestDistance[index] / current_distance);
		correctionVector[2] = p1_to_p2[2] * (1 - GPURestDistance[index] / current_distance);
				
		// Lets make it half that length, so that we can move BOTH p1 and p2.
		
		float correctionVectorHalf[3];
		correctionVectorHalf[0] = correctionVector[0] * 0.5;
		correctionVectorHalf[1] = correctionVector[1] * 0.5;
		correctionVectorHalf[2] = correctionVector[2] * 0.5;
				
				
		// correctionVectorHalf is pointing from p1 to p2, so the length should move p1 half the length needed to satisfy the constraint.
		if(GPUMovableStatus[GPUNeighbourParticlesInddex[index].x])
		{
			//p1->offsetPos(correctionVectorHalf); 
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].x = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].x + correctionVectorHalf[0];
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].y = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].y + correctionVectorHalf[1];
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].z = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].x].z + correctionVectorHalf[2];
		}
				
		if(GPUMovableStatus[GPUNeighbourParticlesInddex[index].y])
		{
			//p2->offsetPos(-correctionVectorHalf); 
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].x = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].x - correctionVectorHalf[0];
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].y = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].y - correctionVectorHalf[1];
			GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].z = GPUParticles_pos_array[GPUNeighbourParticlesInddex[index].y].z - correctionVectorHalf[2];
		}
	
	
	
	
	}
	//__syncthreads();
	
}




void timeStepCudaKernel(float3* GPUParticles_pos_array,int2* GPUNeighbourParticlesInddex,float* GPURestDistance,bool* GPUMovableStatus, int NoOfwidthparticles,int NoOfheightparticle, int ToatalThreads)
{

	dim3 DimBlock=dim3(NoOfwidthparticles,1,1);
	dim3 DimGrid=dim3(NoOfheightparticle, 1, 1);
	
	
	timeStep_Kernel<<<DimGrid, DimBlock>>>(GPUParticles_pos_array,GPUNeighbourParticlesInddex,GPURestDistance,GPUMovableStatus,ToatalThreads);
	
}
	

__global__ void timeStepDisplacement_Kernel( float3* KParticles_pos_array,float3* KParticles_old_pos_array,bool* GPUMovableStatus,float dampingFactor, float3* partcleArray,int particle_width, int particle_height)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	dampingFactor = 0.000f;
	if(GPUMovableStatus[index])
	{
		float temp[3];
		temp[0] = KParticles_pos_array[index].x;
		temp[1] = KParticles_pos_array[index].y;
		temp[2] = KParticles_pos_array[index].z;
		
		KParticles_pos_array[index].x = KParticles_pos_array[index].x + (KParticles_pos_array[index].x - KParticles_old_pos_array[index].x)*(1.0 - dampingFactor) + partcleArray[index].x * TIME_STEPSIZE2;
		KParticles_pos_array[index].y = KParticles_pos_array[index].y + (KParticles_pos_array[index].y - KParticles_old_pos_array[index].y)*(1.0 - dampingFactor) + partcleArray[index].y * TIME_STEPSIZE2;
		KParticles_pos_array[index].z = KParticles_pos_array[index].z + (KParticles_pos_array[index].z - KParticles_old_pos_array[index].z)*(1.0 - dampingFactor) + partcleArray[index].z * TIME_STEPSIZE2;
				
		KParticles_old_pos_array[index].x = temp[0];
		KParticles_old_pos_array[index].y = temp[1];
		KParticles_old_pos_array[index].z = temp[2];
		
		partcleArray[index].x = 0.0f;
		partcleArray[index].y = 0.0f;
		partcleArray[index].z = 0.0f;
				
	}
}



void timeStepDisplacementCudaKernel(float3* KParticles_pos_array,float3* KParticles_old_pos_array,bool* GPUMovableStatus,float dampingFactor, float3* partcleArray,int particle_width, int particle_height)
{

	dim3 DimBlock=dim3(particle_width,1,1);
	dim3 DimGrid=dim3(particle_height, 1, 1);
	
	timeStepDisplacement_Kernel<<<DimGrid, DimBlock>>>(KParticles_pos_array,KParticles_old_pos_array,GPUMovableStatus,dampingFactor, partcleArray,particle_width, particle_height);

}


__global__ void calculateNormal_Kernel(float3* KParticles_pos_array,float3* KParticles_Normal, int Kparticle_width, int Kparticle_height)
{

	int thread_no = blockIdx.x * blockDim.x + threadIdx.x;
	
	
	float *normal;
	float3 TrianglePoint1, TrianglePoint2, TrianglePoint3;
	float v1[3], v2[3];
	
	if(((thread_no) < ((Kparticle_width * Kparticle_height) - blockDim.x  )) && ((thread_no % blockDim.x) < (blockDim.x-1)))
	{
		
		TrianglePoint1 = KParticles_pos_array[thread_no ];  //getParticle(x+1,y);
		TrianglePoint2 = KParticles_pos_array[thread_no + blockDim.x];  //getParticle(x,y);
		TrianglePoint3 = KParticles_pos_array[thread_no + 1];   //getParticle(x,y+1);
			
			
		v1[0] = TrianglePoint2.x - TrianglePoint1.x;
		v1[1] = TrianglePoint2.y - TrianglePoint1.y;
		v1[2] = TrianglePoint2.z - TrianglePoint1.z;
			
		v2[0] = TrianglePoint3.x - TrianglePoint1.x;
		v2[1] = TrianglePoint3.y - TrianglePoint1.y;
		v2[2] = TrianglePoint3.z - TrianglePoint1.z;
			
		normal = GPUcross(&v1[0], &v2[0]);
		
		int particalIndex = 0;
		particalIndex = thread_no  ;
			
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
		
		particalIndex = thread_no + blockDim.x;
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
			
		particalIndex = thread_no+1;	
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
			
			
		normal = NULL;
		//TrianglePoint1 = NULL; TrianglePoint2 = NULL; TrianglePoint3=NULL;
		v1[0] = 0.0f; v1[1] = 0.0f; v1[2] = 0.0f;
		v2[0] = 0.0f; v2[1] = 0.0f; v2[2] = 0.0f;
		


		TrianglePoint1 = KParticles_pos_array[thread_no + 1];  //getParticle(x+1,y+1);
		TrianglePoint2 = KParticles_pos_array[thread_no+ blockDim.x ];  //getParticle(x+1,y);
		TrianglePoint3 = KParticles_pos_array[thread_no + blockDim.x + 1];   //getParticle(x,y+1);
		
					
		v1[0] = TrianglePoint2.x - TrianglePoint1.x;
		v1[1] = TrianglePoint2.y - TrianglePoint1.y;
		v1[2] = TrianglePoint2.z - TrianglePoint1.z;
			
		v2[0] = TrianglePoint3.x - TrianglePoint1.x;
		v2[1] = TrianglePoint3.y - TrianglePoint1.y;
		v2[2] = TrianglePoint3.z - TrianglePoint1.z;
			
		normal = GPUcross(&v1[0], &v2[0]);
		
		particalIndex = thread_no  + 1;
			
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
		
		particalIndex =  thread_no + blockDim.x;
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
			
		particalIndex = thread_no + blockDim.x+ 1 ;	
		KParticles_Normal[particalIndex].x = normal[0];
		KParticles_Normal[particalIndex].y = normal[1];
		KParticles_Normal[particalIndex].z = normal[2];
			
	
	}

}

void calculateNormalCudaKernel(float3* KParticles_pos_array,float3* KParticles_Normal, int Kparticle_width, int Kparticle_height)
{
	dim3 DimBlock=dim3( Kparticle_width ,1,1);
	dim3 DimGrid=dim3( Kparticle_height, 1, 1);

	calculateNormal_Kernel<<<DimGrid, DimBlock>>>(KParticles_pos_array, KParticles_Normal, Kparticle_width, Kparticle_height);
}	


__global__  void render_Kernel(float3* KtriangleVertices,float3* KtriangleVertices_normal, float3* KParticles_pos_array, float3* KParticles_Normal, int Kparticle_width, int Kparticle_height)
{

	int thread_no = blockIdx.x * blockDim.x + threadIdx.x;
	
	int index = thread_no * 6;
	
	
	if(((thread_no) < ((Kparticle_width * Kparticle_height) - blockDim.x  )) && ((thread_no % blockDim.x) < (blockDim.x-1)))
	{
	float3 point = KParticles_pos_array[ thread_no  ]; //getParticle(x, y + 1);
	float3 pointNormal = KParticles_Normal[ thread_no  ]; //getParticleNormals(x, y+1);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	index++;
	
	point = KParticles_pos_array[ thread_no + blockDim.x ]; //getParticle(x, y);
	pointNormal = KParticles_Normal[ thread_no + blockDim.x  ]; //getParticleNormals(x, y);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	index++;
	
	
	point = KParticles_pos_array[ thread_no + 1 ]; //getParticle(x + 1, y);
	pointNormal = KParticles_Normal[ thread_no + 1 ]; //getParticleNormals(x + 1, y);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	index++;
	
	point = KParticles_pos_array[ thread_no + 1 ]; //getParticle(x, y + 1);
	pointNormal = KParticles_Normal[ thread_no + 1 ]; //getParticleNormals(x, y + 1);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	index++;
	
	point = KParticles_pos_array[ thread_no + blockDim.x  ]; //getParticle(x + 1, y);
	pointNormal = KParticles_Normal[ thread_no + blockDim.x ]; //getParticleNormals(x + 1, y);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	index++;
	
	point = KParticles_pos_array[ thread_no + blockDim.x + 1 ]; //getParticle(x + 1, y);
	pointNormal = KParticles_Normal[ thread_no + blockDim.x + 1 ]; //getParticleNormals(x + 1, y);
	
	
	KtriangleVertices[index].x = point.x;
	KtriangleVertices[index].y = point.y;
	KtriangleVertices[index].z = point.z;
	
	KtriangleVertices_normal[index].x = pointNormal.x;
	KtriangleVertices_normal[index].y = pointNormal.y;
	KtriangleVertices_normal[index].z = pointNormal.z;
	
	//index++;
	}

}

void renderCudaKernel(float3* KtriangleVertices,float3* KtriangleVertices_normal, float3* KParticles_pos_array, float3* KParticles_Normal, int Kparticle_width, int Kparticle_height)
{
	dim3 DimBlock=dim3( Kparticle_width ,1,1);
	dim3 DimGrid=dim3( Kparticle_height, 1, 1);

	render_Kernel<<<DimGrid, DimBlock>>>(KtriangleVertices, KtriangleVertices_normal, KParticles_pos_array, KParticles_Normal, Kparticle_width, Kparticle_height);

}
	